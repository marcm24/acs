#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <chrono>

// kernel for matrix multiplication
__global__ void matrixMultiplyKernel(const int* A, const int* B, int* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        int sum = 0;
        for (int k = 0; k < N; ++k) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

// helper function to handle CUDA errors
void checkCudaError(hipError_t status, const char* message) {
    if (status != hipSuccess) {
        fprintf(stderr, "%s: %s\n", message, hipGetErrorString(status));
        exit(EXIT_FAILURE);
    }
}

int main() {
    const int N = 3000; // matrix dimension
    const int size = N * N * sizeof(int);

    // allocate memory for host matrices
    int* A = (int*)malloc(size);
    int* B = (int*)malloc(size);
    int* C = (int*)malloc(size);

    // initialize matrices A and B with random values
    for (int i = 0; i < N * N; ++i) {
        A[i] = rand() % 10;
        B[i] = rand() % 10;
    }

    // allocate memory for device matrices
    int* d_A, * d_B, * d_C;
    checkCudaError(hipMalloc((void**)&d_A, size), "Failed to allocate device memory for A");
    checkCudaError(hipMalloc((void**)&d_B, size), "Failed to allocate device memory for B");
    checkCudaError(hipMalloc((void**)&d_C, size), "Failed to allocate device memory for C");

    // copy matrices A and B to device memory
    checkCudaError(hipMemcpy(d_A, A, size, hipMemcpyHostToDevice), "Failed to copy A to device");
    checkCudaError(hipMemcpy(d_B, B, size, hipMemcpyHostToDevice), "Failed to copy B to device");

    // Define block and grid sizes
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // measure the execution time of the kernel
    auto start = std::chrono::high_resolution_clock::now();

    // launch the kernel
    matrixMultiplyKernel << <blocksPerGrid, threadsPerBlock >> > (d_A, d_B, d_C, N);

    // synchronize to ensure kernel execution is complete
    checkCudaError(hipDeviceSynchronize(), "Kernel execution failed");

    auto stop = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> duration = stop - start;

    // copy result matrix C from device to host
    checkCudaError(hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost), "Failed to copy C to host");

    // output the result matrix and execution time
  /*  printf("Result Matrix (C):\n");
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            printf("%d ", C[i * N + j]);
        }
        printf("\n");
    }*/
    printf("\nTime taken for matrix multiplication of size %d: %.3f ms\n", N, duration.count());

    // free device and host memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(A);
    free(B);
    free(C);

    return 0;
}
