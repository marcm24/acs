#include "hip/hip_runtime.h"


#include <stdio.h>
#include <cstdlib>
#include <chrono>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    const int arraySize = 100; // change this to set the size of the vectors
    int *a = new int[arraySize];
    int *b = new int[arraySize];
    int *c = new int[arraySize]{0};

    // fill vectors with random values
    for (int i = 0; i < arraySize; ++i) {
        a[i] = rand() % 100; // Random values between 0 and 99
        b[i] = rand() % 100; // Random values between 0 and 99
    }

    // start timing
    auto start = std::chrono::high_resolution_clock::now();

    // add vectors in parallel
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    // end timing
    auto stop = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> duration = stop - start;

    // output the result for small-sized vectors (optional)
    if (arraySize <= 10) { // Limit output to small sizes for readability
        printf("{");
        for (int i = 0; i < arraySize; ++i) {
            printf("%d%s", a[i], i < arraySize - 1 ? "," : "");
        }
        printf("} + {");
        for (int i = 0; i < arraySize; ++i) {
            printf("%d%s", b[i], i < arraySize - 1 ? "," : "");
        }
        printf("} = {");
        for (int i = 0; i < arraySize; ++i) {
            printf("%d%s", c[i], i < arraySize - 1 ? "," : "");
        }
        printf("}\n");
    } else {
        printf("Vector addition completed for size: %d\n", arraySize);
    }

    // output the time taken
    printf("Time taken for addition: %.6f ms\n", duration.count());

    // Cleanup
    delete[] a;
    delete[] b;
    delete[] c;

    return 0;
}

// helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // allocate GPU buffers for three vectors (two input, one output).
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
