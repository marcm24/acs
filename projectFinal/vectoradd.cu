#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cstdlib>
#include <chrono>

hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size);

__global__ void addKernel(int* c, const int* a, const int* b, int size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Compute global thread index
    if (i < size) { // Ensure we don't access out-of-bounds memory
        c[i] = a[i] + b[i];
    }
}

int main()
{
    const int arraySize = 1000000; 
    int* a = new int[arraySize];
    int* b = new int[arraySize];
    int* c = new int[arraySize] {0};

    // fill vectors with random values
    for (int i = 0; i < arraySize; ++i) {
        a[i] = rand() % 100; // Random values between 0 and 99
        b[i] = rand() % 100; // Random values between 0 and 99
    }

    // start timing
    auto start = std::chrono::high_resolution_clock::now();

    // add vectors in parallel
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    // end timing
    auto stop = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> duration = stop - start;

    printf("Vector addition completed for size: %d\n", arraySize);
    printf("Time taken for addition: %.6f ms\n", duration.count());

    // Cleanup
    delete[] a;
    delete[] b;
    delete[] c;

    return 0;
}

hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size)
{
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;
    hipError_t cudaStatus;

    // choose which GPU to run on
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // copy input vectors from host to device
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // launch kernel with multiple blocks and threads
    int threadsPerBlock = 256; // Typical value
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock; // Ceiling division
    addKernel << <blocksPerGrid, threadsPerBlock >> > (dev_c, dev_a, dev_b, size);

    // check for errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // wait for the kernel to finish
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // copy result from device to host
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}