#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <chrono> // for timing
#include <stdlib.h> // for rand()

// helper function for using CUDA to perform 2D convolution on a matrix
hipError_t convolutionWithCuda(int* output, const int* input, const int* kernel, int matrixSize, int kernelSize);

// CUDA kernel for performing 2D convolution
__global__ void convolutionKernel(int* output, const int* input, const int* kernel, int matrixSize, int kernelSize)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < matrixSize && j < matrixSize) {
        int offset = kernelSize / 2;
        int sum = 0;

        // Perform convolution operation
        for (int m = -offset; m <= offset; ++m) {
            for (int n = -offset; n <= offset; ++n) {
                int row = i + m;
                int col = j + n;
                if (row >= 0 && row < matrixSize && col >= 0 && col < matrixSize) {
                    sum += input[row * matrixSize + col] * kernel[(m + offset) * kernelSize + (n + offset)];
                }
            }
        }
        output[i * matrixSize + j] = sum;
    }
}

int main()
{
    int matrixSize = 3000; // set the matrix size
    int kernelSize = 5;    // set the kernel size (odd numbers like 3, 5, 7 are typical)

    // define input matrix and kernel
    int* input = (int*)malloc(matrixSize * matrixSize * sizeof(int));
    int* kernel = (int*)malloc(kernelSize * kernelSize * sizeof(int));
    int* output = (int*)malloc(matrixSize * matrixSize * sizeof(int));

    // initialize input matrix with random values
    for (int i = 0; i < matrixSize * matrixSize; ++i) {
        input[i] = rand() % 100; // Random values between 0 and 99
    }

    // initialize kernel matrix with random values
    for (int i = 0; i < kernelSize * kernelSize; ++i) {
        kernel[i] = rand() % 3; // Random values between 0 and 2 for simplicity
    }

    // start timing
    auto start = std::chrono::high_resolution_clock::now();

    // perform convolution using CUDA
    hipError_t cudaStatus = convolutionWithCuda(output, input, kernel, matrixSize, kernelSize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "convolutionWithCuda failed!\n");
        return 1;
    }

    // end timing
    auto stop = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> duration = stop - start;

    // print output matrix for small sizes (optional)
    if (matrixSize <= 10) {
        printf("Input Matrix:\n");
        for (int i = 0; i < matrixSize; ++i) {
            for (int j = 0; j < matrixSize; ++j) {
                printf("%d ", input[i * matrixSize + j]);
            }
            printf("\n");
        }
        printf("\nOutput Matrix (after convolution):\n");
        for (int i = 0; i < matrixSize; ++i) {
            for (int j = 0; j < matrixSize; ++j) {
                printf("%d ", output[i * matrixSize + j]);
            }
            printf("\n");
        }
    }
    else {
        printf("Convolution completed for matrix size: %d\n", matrixSize);
    }

    // Output the time taken for convolution
    printf("Time taken for 2D convolution: %f ms\n", duration.count());

    // Free memory
    free(input);
    free(kernel);
    free(output);

    return 0;
}

// helper function for using CUDA to perform 2D convolution
hipError_t convolutionWithCuda(int* output, const int* input, const int* kernel, int matrixSize, int kernelSize)
{
    int* dev_input = 0;
    int* dev_kernel = 0;
    int* dev_output = 0;
    hipError_t cudaStatus;

    // choose which GPU to run on (change this on a multi-GPU system)
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?\n");
        return cudaStatus;
    }

    // allocate GPU buffers for input, kernel, and output matrices
    cudaStatus = hipMalloc((void**)&dev_input, matrixSize * matrixSize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed for input matrix!\n");
        return cudaStatus;
    }

    cudaStatus = hipMalloc((void**)&dev_kernel, kernelSize * kernelSize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed for kernel!\n");
        return cudaStatus;
    }

    cudaStatus = hipMalloc((void**)&dev_output, matrixSize * matrixSize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed for output matrix!\n");
        return cudaStatus;
    }

    // copy input and kernel matrices from host memory to GPU buffers
    cudaStatus = hipMemcpy(dev_input, input, matrixSize * matrixSize * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed for input matrix!\n");
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(dev_kernel, kernel, kernelSize * kernelSize * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed for kernel matrix!\n");
        return cudaStatus;
    }

    // launch kernel with grid and block sizes based on matrix size
    dim3 blockSize(16, 16);
    dim3 gridSize((matrixSize + blockSize.x - 1) / blockSize.x, (matrixSize + blockSize.y - 1) / blockSize.y);

    convolutionKernel << <gridSize, blockSize >> > (dev_output, dev_input, dev_kernel, matrixSize, kernelSize);

    // check for errors during kernel launch
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "convolutionKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return cudaStatus;
    }

    // synchronize device
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching convolutionKernel!\n", cudaStatus);
        return cudaStatus;
    }

    // copy output matrix from GPU buffer to host memory
    cudaStatus = hipMemcpy(output, dev_output, matrixSize * matrixSize * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed for output matrix!\n");
        return cudaStatus;
    }

    // free GPU memory
    hipFree(dev_output);
    hipFree(dev_input);
    hipFree(dev_kernel);

    return hipSuccess;
}
